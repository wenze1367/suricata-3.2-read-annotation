
#include <hip/hip_runtime.h>
/* Copyright (C) 2007-2012 Open Information Security Foundation
 *
 * You can copy, redistribute or modify this Program under the terms of
 * the GNU General Public License version 2 as published by the Free
 * Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * version 2 along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA
 * 02110-1301, USA.
 */

/**
 * \file
 *
 * \author Anoop Saldanha <anoopsaldanha@gmail.com>
 *
 * The Cuda kernel for MPM AC.
 *
 * \todo - This is a basic version of the kernel.
 *       - Support 16 bit state tables.
 *       - Texture memory.
 *       - Multiple threads per blocks of threads.  Make use of
 *         shared memory/texture memory.
 */

extern "C"
__global__ void SCACCudaSearch64(unsigned char *d_buffer,
                                 unsigned int d_buffer_start_offset,
                                 unsigned int *o_buffer,
                                 unsigned int *results_buffer,
                                 unsigned int nop,
                                 unsigned char *tolower)
{
    unsigned int u = 0;
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= nop)
        return;

    unsigned int buflen = *((unsigned long *)(d_buffer + (o_buffer[tid] - d_buffer_start_offset)));
    unsigned int (*state_table_u32)[256] =
        (unsigned int (*)[256])*((unsigned long *)(d_buffer + (o_buffer[tid] - d_buffer_start_offset) + 8));
    unsigned char *buf = (d_buffer + (o_buffer[tid] - d_buffer_start_offset) + 16);

    unsigned int state = 0;
    unsigned int matches = 0;
    unsigned int *results = (results_buffer + ((o_buffer[tid] - d_buffer_start_offset) * 2) + 1);
    for (u = 0; u < buflen; u++) {
        state = state_table_u32[state & 0x00FFFFFF][tolower[buf[u]]];
        if (state & 0xFF000000) {
            results[matches++] = u;
            results[matches++] = state & 0x00FFFFFF;
        }
    }

    *(results - 1) = matches;
    return;
}

extern "C"
__global__ void SCACCudaSearch32(unsigned char *d_buffer,
                                 unsigned int d_buffer_start_offset,
                                 unsigned int *o_buffer,
                                 unsigned int *results_buffer,
                                 unsigned int nop,
                                 unsigned char *tolower)
{
    unsigned int u = 0;
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= nop)
        return;

    unsigned int buflen = *((unsigned int *)(d_buffer + (o_buffer[tid] - d_buffer_start_offset)));
    unsigned int (*state_table_u32)[256] =
        (unsigned int (*)[256])*((unsigned int *)(d_buffer + (o_buffer[tid] - d_buffer_start_offset) + 4));
    unsigned char *buf = (d_buffer + (o_buffer[tid] - d_buffer_start_offset) + 8);

    unsigned int state = 0;
    unsigned int matches = 0;
    unsigned int *results = (results_buffer + ((o_buffer[tid] - d_buffer_start_offset) * 2) + 1);
    for (u = 0; u < buflen; u++) {
        state = state_table_u32[state & 0x00FFFFFF][tolower[buf[u]]];
        if (state & 0xFF000000) {
            results[matches++] = u;
            results[matches++] = state & 0x00FFFFFF;
        }
    }

    *(results - 1) = matches;
    return;
}
